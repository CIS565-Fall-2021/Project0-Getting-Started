#include "hip/hip_runtime.h"
/**
 * @file
 * @brief     CIS 565 Version Checker
 * @details   A simple CUDA hello-world-style program for Patrick Cozzi's
 *            CIS 565: GPU Programming, at the University of Pennsylvania.
 * @authors   Starter code: Yining Karl Li, Liam Boone, Harmony Li, Kai Ninomiya
 * @copyright University of Pennsylvania
 */

#include <cstdio>
#include <hip/hip_runtime_api.h>
#include <>
#include "kernel.h"

void checkCUDAError(const char *msg) {
    hipError_t err = hipGetLastError();
    if (hipSuccess != err) {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

/**
 * Ver  Color           HEX         RGB
 * 0  -> White          #ffffff     rgb(255, 255, 255)
 * 1  -> Red            #ff0000     rgb(255,   0,   0)
 * 2  -> Orange         #ff8000     rgb(255, 128,   0)
 * 3  -> Yellow         #ffff00     rgb(255, 255,   0)
 * 4  -> Lime           #80ff00     rgb(128, 255,   0)
 * 5  -> Green          #00ff00     rgb(  0, 255,   0)
 * 6  -> Spring Green   #00ff80     rgb(  0, 255, 128)
 * 7  -> Cyan           #00ffff     rgb(  0, 255, 255)
 * 8  -> Dodger Blue    #0080ff     rgb(  0, 128, 255)
 * 9  -> Blue           #0000ff     rgb(  0,   0, 255)
 * 10 -> Purple         #8000ff     rgb(128,   0, 255)
 * 11 -> Violet         #ff00ff     rgb(255,   0, 255)
 * 12 -> Magenta        #ff0080     rgb(255,   0, 128)
 * *  -> Black          #000000     rgb(  0,   0,   0)
 */
__host__ __device__ void versionToColor(uchar4* pixel, int version) {
    switch(version) {
        case 0:
            pixel->x = 255; pixel->y = 255; pixel->z = 255; break;
        case 1:
            pixel->x = 255; pixel->y =   0; pixel->z =   0; break;
        case 2:
            pixel->x = 255; pixel->y = 128; pixel->z =   0; break;
        case 3:
            pixel->x = 255; pixel->y = 255; pixel->z =   0; break;
        case 4:
            pixel->x = 128; pixel->y = 255; pixel->z =   0; break;
        case 5:
            pixel->x =   0; pixel->y = 255; pixel->z =   0; break;
        case 6:
            pixel->x =   0; pixel->y = 255; pixel->z = 128; break;
        case 7:
            pixel->x =   0; pixel->y = 255; pixel->z = 255; break;
        case 8:
            pixel->x =   0; pixel->y = 128; pixel->z = 255; break;
        case 9:
            pixel->x =   0; pixel->y =   0; pixel->z = 255; break;
        case 10:
            pixel->x = 128; pixel->y =   0; pixel->z = 255; break;
        case 11:
            pixel->x = 255; pixel->y =   0; pixel->z = 255; break;
        case 12:
            pixel->x = 255; pixel->y =   0; pixel->z = 128; break;
        default:
            pixel->x =   0; pixel->y =   0; pixel->z =   0;
    }
}

// Kernel that writes the image to the OpenGL PBO directly.
__global__ void createVersionVisualization(uchar4* PBOpos, int width, int height, int major, int minor) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int index = x + (y * width);

    if (x <= width && y <= height) {
        // Each thread writes one pixel location in the texture (textel)
        PBOpos[index].w = 0;
        PBOpos[index].x = 0;
        PBOpos[index].y = 0;
        PBOpos[index].z = 0;

        int ver = y < height / 2 ? major : minor;
        versionToColor(&PBOpos[index], ver);
    }
}

// Wrapper for the __global__ call that sets up the kernel calls
void kernelVersionVis(uchar4* PBOpos, int width, int height, int major, int minor) {
    // set up crucial magic
    unsigned int blockSize = 16;
    dim3 threadsPerBlock(blockSize, blockSize);

    unsigned int blocksX = (width + blockSize - 1) / blockSize;
    unsigned int blocksY = (height + blockSize - 1) / blockSize;
    dim3 fullBlocksPerGrid(blocksX, blocksY);

    //kernel launches
    createVersionVisualization <<< fullBlocksPerGrid, threadsPerBlock>>>(PBOpos, width, height, major, minor);

    // make certain the kernel has completed
    hipDeviceSynchronize();

    checkCUDAError("Kernel failed!");
}
